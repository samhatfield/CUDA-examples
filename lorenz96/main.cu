#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>

// Dimension of model
__constant__ int N;

// Timestep
__constant__ PREC dt;

// Forcing
__constant__ PREC F;

// ODE device function
__device__ PREC dXdT(PREC x__2, PREC x__1, PREC x, PREC x_1) {
    return (x_1 - x__2)*x__1 - x + F;
}

// Index shifter device function
__device__ int shft(int n, int m) {
    return (n + m + N)%N;
}

// Main step kernel function
__global__ void step(PREC* __restrict__ in, PREC* __restrict__ out) {
    // Get global thread ID
    int tid = threadIdx.x + blockDim.x*blockIdx.x;

    // Intermediate steps
    PREC k1, k2;

    if (tid < N) {
        // Compute k1
        k1 = dXdT(in[shft(tid,-2)], in[shft(tid,-1)], in[tid], in[shft(tid,1)]);

        // Add h*k1 to step
        in[tid] += dt*k1;

        __syncthreads();

        // Compute k2
        k2 = dXdT(in[shft(tid,-2)], in[shft(tid,-1)], in[tid], in[shft(tid,1)]);

        // Get local state
        out[tid] = in[tid] + 0.5*dt*(k2 - k1);
    }
}

int main(int argc, const char **argv) {
    // Simulation parameters
    int h_N = 40;
    PREC h_dt = 0.05;
    PREC h_F = 10.0;
    int length = 1000;

    // Storage vectors
    PREC *h_state, *h_hist, *d_prev, *d_next, *d_temp;

    // Kernel parameters
    int nThreadsPerBlock = 64;
    int nBlocks = 1 + ((h_N - 1)/nThreadsPerBlock);

    // Initialise card
    findCudaDevice(argc, argv);

    // Move global constants to device
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(h_N)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dt), &h_dt, sizeof(h_dt)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(F), &h_F, sizeof(h_F)));

    // Allocate memory on host and device
    h_state = (PREC *)malloc(sizeof(PREC)*h_N);
    h_hist  = (PREC *)malloc(sizeof(PREC)*length);

    checkCudaErrors(hipMalloc((void **)&d_prev, sizeof(PREC)*h_N));
    checkCudaErrors(hipMalloc((void **)&d_next, sizeof(PREC)*h_N));

    // Set initial conditions
    for (int i = 0; i < h_N; i++) {
        h_state[i] = (PREC)rand()/RAND_MAX;
    }

    printf("%f %f\n", h_state[0], h_state[1]);

    // Copy initial conditions to device
    checkCudaErrors(hipMemcpy(d_prev, h_state, sizeof(PREC)*h_N, hipMemcpyHostToDevice));

    // Set initial condition in history array
    h_hist[0] = h_state[0];

    // Run forecast
    printf("Running forecast with %d blocks and %d threads per block\n", nBlocks, nThreadsPerBlock);
    for (int i = 1; i < length; i++) {
        // Step forward once
        step<<<nBlocks, nThreadsPerBlock>>>(d_prev, d_next);
        getLastCudaError("step execution failed\n");

        // Store one variable
        checkCudaErrors(hipMemcpy(&h_hist[i], &d_next[0], sizeof(PREC), hipMemcpyDeviceToHost));

        printf("%f\n", h_hist[i]);

        // Swap prev and next pointers
        d_temp = d_prev; d_prev = d_next; d_next = d_temp;
    }

    // Copy back results
    checkCudaErrors(hipMemcpy(h_state, d_next, sizeof(PREC)*h_N,hipMemcpyDeviceToHost));

    // Free up memory
    free(h_state);
    free(h_hist);
    checkCudaErrors(hipFree(d_prev));
    checkCudaErrors(hipFree(d_next));

    hipDeviceReset();
}
